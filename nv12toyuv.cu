#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "nv12toyuv.h"
__global__ void kernel_yuvconv(uint8_t* src, uint8_t* dst1, uint8_t* dst2)
{
	int i = threadIdx.x;
	dst1[i] = src[2 * i];
	dst2[i] = src[2 * i + 1];
}

void NV12ToYUV(uint8_t* src, uint8_t* des_u, uint8_t* des_v, int width, int height,int pitch)
{
    //pitch = 1920     width = 2048
	uint8_t *src_uv, *mid_uv;
	int width_2 = width / 2;//width_2 = 1024
	int width_4 = width / 4;//width_4 = 512
	src_uv = src + pitch * height;
	hipMalloc((void**)&mid_uv, width);
	for (int i = 0; i < height / 2; i ++)
	{
		hipMemcpy(mid_uv,  src_uv,  pitch, hipMemcpyDeviceToDevice);//pitch = 1920

		if (width_2 >= 512)
		{
			kernel_yuvconv<<<1, width_4>>>(mid_uv, des_u, des_v);//一个网格，512个线程块，width_4这里最好设置为32的倍数
			kernel_yuvconv<<<1, width_4>>>(mid_uv + width_2, des_u + width_4, des_v + width_4);
		}
		des_u	+= width_2;
		des_v	+= width_2;
		src_uv  += pitch;
	}
	hipFree(mid_uv);
}
